/****************************************************************************************
*                                                                                       *
*    TestingC CUDA Utility Functions                                                    *
*                                                                                       *
*    This file contains tests for all the C CUDA funtions available in cuda_utils.h:    *
*        - Get information about available CUDA enabled GPU(s)                          *
*        - Create Timing Events                                                         *
*        - Allocate memory on GPU(s)                                                    *
*        - Do some Work in GPU                                                          *
*        - Free memory on GPU(s)                                                        *
*        - Copy data to/from GPU(s)                                                     *
*        - Destroy Timing Events                                                        *
*                                                                                       *
*                                                                                       *
*    (C) Christos Kannas                                                                *
*    The Cyprus Institute                                                               *
*    mailto: c.kannas@cyi.ac.cy                                                         *
*                                                                                       *
****************************************************************************************/
/**
* Testing Functions in cuda_utils.h
*/

#include "cuda_utils.h"

#define N 10

int main() 
{
    /* Device specific variables */
    static int devices = -1;
    static size_t *chunk;      // Memory block required per CUDA device based on memory requirements
    static size_t *chunk32;    // Memory block, rounded to nearest multiple of 32, required per CUDA device
    static size_t *stagesize;  // Total memory in bytes required per CUDA device
    /* Data variables */
    int i;
    //float test_array[N], *d_test_array, final_array[N];
    //size_t size = N*sizeof(float);
    
    /* Solver initialization */
    if(devices == -1) {
        displayCudaDevices(&devices);

        chunk = (size_t *)malloc(sizeof(size_t) * devices);
        chunk32 = (size_t *)malloc(sizeof(size_t) * devices);
        stagesize = (size_t *)malloc(sizeof(size_t) * devices);

        /* Divide equaly across available CUDA devices the required ammount of memory */
        /* Estimate overhead in device memory */
        size_t overhead = N * (sizeof(float) + sizeof(int));
        overhead /= devices;

        /* Estimate cell size in bytes */
        size_t cellsize = sizeof(float) * N;
        cellsize /= devices;

        fprintf(stderr, "overhead: %i\n", overhead);
        fprintf(stderr, "cellsize: %i\n\n", cellsize);

        for(i =0; i < devices; i++){
            hipDeviceProp_t props;

            toolTestExec(hipGetDeviceProperties(&props, i));
            /* Estimate number of cells that will fit in device memory */
            chunk[i] = (props.totalGlobalMem - overhead) / cellsize;
            /* Don't exceed device grid limits */
            size_t maxblocks = props.maxGridSize[0];
            if(chunk[i] > maxblocks) 
                chunk[i] = maxblocks;
            /* Round up to next multiple of 32 */
            chunk32[i] = (chunk[i] + (KPP_CUDA_WRAP_SIZE -1)) & ~(KPP_CUDA_WRAP_SIZE -1);
            /* Total Memory required */
            stagesize[i] = cellsize * chunk32[i];

            fprintf(stderr, "device: %i\n", i);
            fprintf(stderr, "chunk: %i\n", chunk[i]);
            fprintf(stderr, "maxblocks: %i\n", maxblocks);
            fprintf(stderr, "chunk32: %i\n", chunk32[i]);
            fprintf(stderr, "stagesize: %i\n\n", stagesize[i]);
        }
    }
    
    /* Allocate write combined, page-locked host memory */
    float **h_test_array = (float **)malloc(sizeof(float *) * devices);
     /* Allocate device memory for species concentrations */
    float **d_test_array = (float **)malloc(sizeof(float *) * devices);
    //
    for(i = 0; i < devices; i++){
        /* Enable device */
        toolTestExec(hipSetDevice(i));

        // Initialize Timing Events
        InitEvents();

        /* Allocate CUDA host memory */
        KPP_CUDA_dump_message("Allocating CUDA host memory\n");
        if( hipSuccess != hipHostAlloc( &(h_test_array[i]), stagesize[i], hipHostMallocWriteCombined ) ){
            /* Fall back to page-locked only */
            KPP_CUDA_dump_message("Can't allocate write combined page-locked host memory.\n");
            if( hipSuccess != hipHostMalloc( &(h_test_array[i]), stagesize[i] ) ){
                /* Fall back to regular malloc */
                KPP_CUDA_dump_message("Can't allocate page-locked host memory.\n");
                h_test_array[i] = (float *)malloc(stagesize[i]);
                if( !(h_test_array[i]) ){
                    KPP_CUDA_dump_message("Failed to allocate host memory.\n");
                    exit(-1);
                }
            }
        }
        /* Allocate CUDA device memory */
        gpuMalloc( (void **) &(d_test_array[i]), stagesize[i] );

        // Destroy Timing Events
        FreeEvents();
    }

    // Init host array
    /*
    for(i = 0; i < N; i++)
        test_array[i] = (float) i*2.5;
    */
    // Initialize Timing Events
    //InitEvents();
    // Allocate GPU Memory
    //gpuMalloc((void **)&d_test_array, size);
    // Copy to GPU
    //copyToGPU(size, test_array, d_test_array);
    // Do some work in GPU
    // Copy from GPU
    //copyFromGPU(size, final_array, d_test_array);
    // Free GPU Memory
    //gpuMemFree(d_test_array);
    // Destroy Timing Events
    //FreeEvents();
    
    /*
    for(i = 0; i < N; i++)
        if(test_array[i] != final_array[i])
            fprintf(stderr, "%i: %f != %f", i, test_array[i], final_array[i]);
    */

    return 0;
}
