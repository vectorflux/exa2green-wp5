#include "hip/hip_runtime.h"
/*-------------------- BEGIN kppa_CUDA_rosenbrock.cu BEGIN --------------------*/
/* @file kppa_CUDA_rosenbrock.cu                                               */
/* @author charlesj                                                            */
/* @date 2015-07-06 14:41:45.062659                                            */
/* @brief Solves the system y' = F(t,y) using a Rosenbrock method              */
/*                                                                             */
/* Solves the system y' = F(t,y) using a Rosenbrock method defined by:         */
/*                                                                             */
/*     G = 1 / (H*gamma) - Jacobian(t0,Y0)                                     */
/*     T_i = t0 + Alpha(i) * H                                                 */
/*     Y_i = Y0 + \sum_{j=1}^{i-1} A(i,j)*K_j                                  */
/*     G * K_i = F(T_i, Y_i) + \sum_{j=1}^S C(i,j)/H * K_j                     */
/*               + gamma(i)*dF/dT(t0, Y0)                                      */
/*     Y1 = Y0 + \sum_{j=1}^S M(j)*K_j                                         */
/*                                                                             */
/* For details on Rosenbrock methods and their implementations:                */
/*     (1) E. Harier and G. Wanner,                                            */
/*         "Solving Ordenary Differential Equations II: stiff and              */
/*         differential-algebraic problems." Computational Mathematics,        */
/*         Springer-Verlag (1996)                                              */
/*     (2) KPP - the Kinetic PreProcessor.                                     */
/*         http://people.cs.vt.edu/~asandu/Software/Kpp/                       */
/*                                                                             */
/* Rosenbrock implementations in both (1) and (2) inspired this code.          */
/* This code presents an interface similar to the KPP implementation           */
/* for compatibility with existing systems.                                    */
/*                                                                             */
/* -- Explanation of integer input parameters:                                 */
/*                                                                             */
/*     idata[0] == 0 : F = F(t,y) Depends on T (non-autonomous).               */
/*              != 0 : F = F(y)   Independent of T (autonomous).               */
/*     idata[1] == 0 : Use all values in tolerance vectors.                    */
/*              != 0 : Use only the first value in the tolerance vectors.      */
/*     idata[2] == 0 : Maximum number of integration steps = 100000.           */
/*              != 0 : Maximum number of integration steps = idata[2].         */
/*     idata[3] == 0 : Method is Ros4.                                         */
/*              == 1 : Method is Ros2.                                         */
/*              == 2 : Method is Ros3.                                         */
/*              == 3 : Method is Ros4.                                         */
/*              == 4 : Method is Rodas3.                                       */
/*              == 5 : Method is Rodas4.                                       */
/*              >= 6 : Error.                                                  */
/*     idata[4] == 0 : Assume tolerance vectors are reasonably valued.         */
/*              != 0 : Check tolerance vectors for unreasonable values.        */
/*                                                                             */
/* -- Explanation of real value input parameters:                              */
/*                                                                             */
/*     rdata[0]: Lower bound on the integration step size.                     */
/*               Default: 0.0                                                  */
/*     rdata[1]: Upper bound on the integration step size.                     */
/*               Default: abs(tend - tstart)                                   */
/*     rdata[2]: Starting value for the integration step size.                 */
/*               Default: minimum step size                                    */
/*     rdata[3]: Lower bound on step decrease factor.                          */
/*               Default: 0.2                                                  */
/*     rdata[4]: Upper bound on step increase factor.                          */
/*               Default: 6.0                                                  */
/*     rdata[5]: Step decrease factor after step rejection.                    */
/*               Default: 0.1                                                  */
/*     rdata[6]: Safety factor in computation of new step size.                */
/*               Default: 0.9                                                  */
/*                                                                             */
/* -- Explanation of integer output parameters:                                */
/*                                                                             */
/*     idata[10]: Number of function evaluations.                              */
/*     idata[11]: Number of Jacobian evaluations.                              */
/*     idata[12]: Number of steps taken.                                       */
/*     idata[13]: Number of accepted steps.                                    */
/*     idata[14]: Number of rejected steps.                                    */
/*     idata[15]: Number of LU decompositions.                                 */
/*     idata[16]: Number of forward/backward substitutions.                    */
/*     idata[17]: Number of singular matrix decompositions.                    */
/*     idata[19]: Integrator exit status.                                      */
/*                Zero indicates success.                                      */
/*                Positive values indicate success with warning.               */
/*                Negative values indicate failure.                            */
/*                                                                             */
/* -- Explanation of real-value output parameters:                             */
/*                                                                             */
/*     rdata[10]: The time corresponding to the computed Y upon return.        */
/*     rdata[11]: The last accepted step before exit.                          */
/*                Use this value as rdata[2] in subsequent runs.               */
/*     rdata[12]: Scaled norm of the error vector on exit.                     */
/*                                                                             */
/* This file was generated by Kppa: http://www.paratools.com/Kppa              */
/*-----------------------------------------------------------------------------*/
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
#include "kppa_CUDA_cu_parameters.h"
#include "kppa_CUDA_blas.h"
#include "kppa_CUDA_rates.h"
#include "kppa_CUDA_function.h"
#include "kppa_CUDA_decomp.h"
#include "kppa_CUDA_solve.h"
#include "kppa_CUDA_jacobian.h"
#include "kppa_CUDA_sparse.h"
#include "kppa_CUDA_rosenbrock.h"




#include <float.h>

/* Minimum time delta */
#define MIN_DELT ((double)10.0 * DBL_EPSILON)


/* Rosenbrock new step solution coefficients in device constant memory */
__constant__ double d_M[6];

/* Rosenbrock error estimation coefficients in device constant memory */
__constant__ double d_E[6];

/* Rosenbrock coefficient matrix A in device constant memory */
__constant__ double d_A[15];

/* Rosenbrock coefficient matrix C in device constant memory */
__constant__ double d_C[15];



/*----------------------------------------------------------------------------*/
/* A two-stage L-stable method of order 2                                     */
/*                                                                            */
/* E. Harier and G. Wanner, "Solving Ordenary Differential Equations II:      */
/* stiff and differential-algebraic problems." Computational Mathematics,     */
/* Springer-Verlag (1996)                                                     */
/*                                                                            */
/* @param[out] name     Method name                                           */
/* @param[out] nStage   Number of method stages                               */
/* @param[out] invLoEst One divided by the estimation of local order          */
/* @param[out] M        Coefficients for new step solution                    */
/* @param[out] E        Coefficients for error estimation                     */
/* @param[out] A        Lower triangular coefficient matrix                   */
/* @param[out] C        Lower triangular coefficient matrix                   */
/* @param[out] alpha    Y at stage i is approx. Y(T + H*Alpha_i)              */
/* @param[out] gamma    Stage i Gamma = sum(gamma[j])                         */
/* @param[out] F        Function evaluation flags                             */
/*----------------------------------------------------------------------------*/
void InitRos2(char ** name, int * nStage, double * invLoEst, double M[], double E[],
        double A[], double C[], double alpha[], double gamma[], char F[])
{
    *name = "Ros2";

    *nStage = 2;

    *invLoEst = 0.5; /* 1 / 2 */

    M[0] = 0.8786796564403575;
    M[1] = 0.2928932188134525;

    E[0] = 0.2928932188134525;
    E[1] = 0.2928932188134525;

    A[0] = 0.585786437626905;

    C[0] = -1.17157287525381;

    alpha[0] = 0.0;
    alpha[1] = 1.0;

    gamma[0] =  1.7071067811865475;
    gamma[1] = -1.7071067811865475;

    F[0] = 1;
    F[1] = 1;
} /* END InitRos2 */


/*----------------------------------------------------------------------------*/
/* A three-stage L-stable method of order 3                                   */
/*                                                                            */
/* E. Harier and G. Wanner, "Solving Ordenary Differential Equations II:      */
/* stiff and differential-algebraic problems." Computational Mathematics,     */
/* Springer-Verlag (1996)                                                     */
/*                                                                            */
/* @param[out] name     Method name                                           */
/* @param[out] nStage   Number of method stages                               */
/* @param[out] invLoEst One divided by the estimation of local order          */
/* @param[out] M        Coefficients for new step solution                    */
/* @param[out] E        Coefficients for error estimation                     */
/* @param[out] A        Lower triangular coefficient matrix                   */
/* @param[out] C        Lower triangular coefficient matrix                   */
/* @param[out] alpha    Y at stage i is approx. Y(T + H*Alpha_i)              */
/* @param[out] gamma    Stage i Gamma = sum(gamma[j])                         */
/* @param[out] F        Function evaluation flags                             */
/*----------------------------------------------------------------------------*/
void InitRos3(char ** name, int * nStage, double * invLoEst, double M[], double E[],
        double A[], double C[], double alpha[], double gamma[], char F[])
{
    /* Method name */
    *name = "Ros3";

    /* Number of stages */
    *nStage = 3;

    /* Inverse estimation of local order: 1/3 */
    *invLoEst = 0.3333333333333333;

    /* Coefficients for new step solution */
    M[0] = 1.0;
    M[1] = 6.1697947043828245592553615689730;
    M[2] = -0.4277225654321857332623837380651;

    /* Coefficients for error estimation */
    E[0] = 0.5;
    E[1] = -2.9079558716805469821718236208017;
    E[2] = 0.2235406989781156962736090927619;

    /* Lower triangular coefficient matrix A */
    A[0] = 1.0;
    A[1] = 1.0;
    A[2] = 0.0;

    /* Lower triangular coefficient matrix C */
    C[0] = -1.0156171083877702091975600115545;
    C[1] = 4.0759956452537699824805835358067;
    C[2] = 9.2076794298330791242156818474003;

    /* Two function evaluations */
    F[0] = 1;
    F[1] = 1;
    F[2] = 0;

    /* Y_stage_i ~ Y( T + H*Alpha_i ) */
    alpha[0] = 0.0;
    alpha[1] = 0.43586652150845899941601945119356;
    alpha[2] = 0.43586652150845899941601945119356;

    /* Gamma_i = \sum_j  gamma_{i,j}  */
    gamma[0] = 0.43586652150845899941601945119356;
    gamma[1] = 0.24291996454816804366592249683314;
    gamma[2] = 2.1851380027664058511513169485832;

} /* END InitRos3 */


/*----------------------------------------------------------------------------*/
/* A four-stage L-stable method of order 4                                    */
/*                                                                            */
/* E. Harier and G. Wanner, "Solving Ordenary Differential Equations II:      */
/* stiff and differential-algebraic problems." Computational Mathematics,     */
/* Springer-Verlag (1996)                                                     */
/*                                                                            */
/* @param[out] name     Method name                                           */
/* @param[out] nStage   Number of method stages                               */
/* @param[out] invLoEst One divided by the estimation of local order          */
/* @param[out] M        Coefficients for new step solution                    */
/* @param[out] E        Coefficients for error estimation                     */
/* @param[out] A        Lower triangular coefficient matrix                   */
/* @param[out] C        Lower triangular coefficient matrix                   */
/* @param[out] alpha    Y at stage i is approx. Y(T + H*Alpha_i)              */
/* @param[out] gamma    Stage i Gamma = sum(gamma[j])                         */
/* @param[out] F        Function evaluation flags                             */
/*----------------------------------------------------------------------------*/
void InitRos4(char ** name, int * nStage, double * invLoEst, double M[], double E[],
        double A[], double C[], double alpha[], double gamma[], char F[])
{
    /* Method name */
    *name = "Ros4";

    /* Number of stages */
    *nStage = 4;

    /* Inverse estimation of local order: 1/4 */
    *invLoEst = 0.25;

    /* Coefficients for new step solution */
    M[0] = 2.255570073418735;
    M[1] = 0.2870493262186792;
    M[2] = 0.4353179431840180;
    M[3] = 1.093502252409163;

    /* Coefficients for error estimation */
    E[0] = -0.2815431932141155;
    E[1] = -0.07276199124938920;
    E[2] = -0.1082196201495311;
    E[3] = -1.093502252409163;

    /* Lower triangular coefficient matrix A */
    A[0] = 2.0;
    A[1] = 1.867943637803922;
    A[2] = 0.2344449711399156;
    A[3] = 1.867943637803922;
    A[4] = 0.2344449711399156;
    A[5] = 0.0;

    /* Lower triangular coefficient matrix C */
    C[0] = -7.137615036412310;
    C[1] =  2.580708087951457;
    C[2] =  0.6515950076447975;
    C[3] = -2.137148994382534;
    C[4] = -0.3214669691237626;
    C[5] = -0.6949742501781779;

    /* Three function evaluations */
    F[0] = 1;
    F[1] = 1;
    F[2] = 1;
    F[3] = 0;

    /* Y_stage_i ~ Y( T + H*Alpha_i ) */
    alpha[0] = 0.0;
    alpha[1] = 1.145640000000000;
    alpha[2] = 0.6552168638155900;
    alpha[3] = 0.6552168638155900;

    /* Gamma_i = \sum_j  gamma_{i,j}  */
    gamma[0] = 0.5728200000000000;
    gamma[1] = -1.769193891319233;
    gamma[2] = 0.7592633437920482;
    gamma[3] = -0.1049021087100450;

} /* END InitRos4 */


/*----------------------------------------------------------------------------*/
/* A four-stage stiffly-stable method of order 4                              */
/*                                                                            */
/* E. Harier and G. Wanner, "Solving Ordenary Differential Equations II:      */
/* stiff and differential-algebraic problems." Computational Mathematics,     */
/* Springer-Verlag (1996)                                                     */
/*                                                                            */
/* @param[out] name     Method name                                           */
/* @param[out] nStage   Number of method stages                               */
/* @param[out] invLoEst One divided by the estimation of local order          */
/* @param[out] M        Coefficients for new step solution                    */
/* @param[out] E        Coefficients for error estimation                     */
/* @param[out] A        Lower triangular coefficient matrix                   */
/* @param[out] C        Lower triangular coefficient matrix                   */
/* @param[out] alpha    Y at stage i is approx. Y(T + H*Alpha_i)              */
/* @param[out] gamma    Stage i Gamma = sum(gamma[j])                         */
/* @param[out] F        Function evaluation flags                             */
/*----------------------------------------------------------------------------*/
void InitRodas3(char ** name, int * nStage, double * invLoEst, double M[], double E[],
        double A[], double C[], double alpha[], double gamma[], char F[])
{
    /* Method name */
    *name = "Rodas3";

    /* Number of stages */
    *nStage = 4;

    /* Inverse estimation of local order: 1/3 */
    *invLoEst = 0.3333333333333333;

    /* Coefficients for new step solution */
    M[0] = 2.0;
    M[1] = 0.0;
    M[2] = 1.0;
    M[3] = 1.0;

    /* Coefficients for error estimation */
    E[0] = 0.0;
    E[1] = 0.0;
    E[2] = 0.0;
    E[3] = 1.0;

    /* Lower triangular coefficient matrix A */
    A[0] = 0.0;
    A[1] = 2.0;
    A[2] = 0.0;
    A[3] = 2.0;
    A[4] = 0.0;
    A[5] = 1.0;

    /* Lower triangular coefficient matrix C */
    C[0] = 4.0;
    C[1] = 1.0;
    C[2] = -1.0;
    C[3] = 1.0;
    C[4] = -1.0;
    C[5] = -2.66666666666667;

    /* Three function evaluations */
    F[0] = 1;
    F[1] = 0;
    F[2] = 1;
    F[3] = 1;

    /* Y_stage_i ~ Y( T + H*Alpha_i ) */
    alpha[0] = 0.0;
    alpha[1] = 0.0;
    alpha[2] = 1.0;
    alpha[3] = 1.0;

    /* Gamma_i = \sum_j  gamma_{i,j}  */
    gamma[0] = 0.5;
    gamma[1] = 1.5;
    gamma[2] = 0.0;
    gamma[3] = 0.0;

} /* END InitRodas3 */


/*----------------------------------------------------------------------------*/
/* A six-stage stiffly-stable method of order 4                               */
/*                                                                            */
/* E. Harier and G. Wanner, "Solving Ordenary Differential Equations II:      */
/* stiff and differential-algebraic problems." Computational Mathematics,     */
/* Springer-Verlag (1996)                                                     */
/*                                                                            */
/* @param[out] name     Method name                                           */
/* @param[out] nStage   Number of method stages                               */
/* @param[out] invLoEst One divided by the estimation of local order          */
/* @param[out] M        Coefficients for new step solution                    */
/* @param[out] E        Coefficients for error estimation                     */
/* @param[out] A        Lower triangular coefficient matrix                   */
/* @param[out] C        Lower triangular coefficient matrix                   */
/* @param[out] alpha    Y at stage i is approx. Y(T + H*Alpha_i)              */
/* @param[out] gamma    Stage i Gamma = sum(gamma[j])                         */
/* @param[out] F        Function evaluation flags                             */
/*----------------------------------------------------------------------------*/
void InitRodas4(char ** name, int * nStage, double * invLoEst, double M[], double E[],
        double A[], double C[], double alpha[], double gamma[], char F[])
{
    /* Method name */
    *name = "Rodas4";

    /* Number of stages */
    *nStage = 6;

    /* Inverse estimation of local order: 1/4 */
    *invLoEst = 0.25;

    /* Coefficients for new step solution */
    M[0] = 1.544000000000000;
    M[1] = 6.019134481288629;
    M[2] = 12.53708332932087;
    M[3] = -0.6878860361058950;
    M[4] = 1.0;
    M[5] = 1.0;

    /* Coefficients for error estimation */
    E[0] = 0.0;
    E[1] = 0.0;
    E[2] = 0.0;
    E[3] = 0.0;
    E[4] = 0.0;
    E[5] = 1.0;

    /* Lower triangular coefficient matrix A */
    A[0] = 1.544000000000000;
    A[1] = 0.9466785280815826;
    A[2] = 0.2557011698983284;
    A[3] = 3.314825187068521;
    A[4] = 2.896124015972201;
    A[5] = 0.9986419139977817;
    A[6] = 1.221224509226641;
    A[7] = 6.019134481288629;
    A[8] = 12.53708332932087;
    A[9] = -0.6878860361058950;
    A[10] = 1.221224509226641;
    A[11] = 6.019134481288629;
    A[12] = 12.53708332932087;
    A[13] = -0.6878860361058950;
    A[14] = 1.0;

    /* Lower triangular coefficient matrix C */
    C[0] = -5.668800000000000;
    C[1] = -2.430093356833875;
    C[2] = -0.2063599157091915;
    C[3] = -0.1073529058151375;
    C[4] = -9.594562251023355;
    C[5] = -20.47028614809616;
    C[6] = 7.496443313967647;
    C[7] = -10.24680431464352;
    C[8] = -33.99990352819905;
    C[9] = 11.70890893206160;
    C[10] = 8.083246795921522;
    C[11] = -7.981132988064893;
    C[12] = -31.52159432874371;
    C[13] = 16.31930543123136;
    C[14] = -6.058818238834054;

    /* Six function evaluations */
    F[0] = 1;
    F[1] = 1;
    F[2] = 1;
    F[3] = 1;
    F[4] = 1;
    F[5] = 1;

    /* Y_stage_i ~ Y( T + H*Alpha_i ) */
    alpha[0] = 0.000;
    alpha[1] = 0.386;
    alpha[2] = 0.210;
    alpha[3] = 0.630;
    alpha[4] = 1.000;
    alpha[5] = 1.000;

    /* Gamma_i = \sum_j  gamma_{i,j}  */
    gamma[0] = 0.2500000000000000;
    gamma[1] = -0.1043000000000000;
    gamma[2] = 0.1035000000000000;
    gamma[3] = -0.03620000000000023;
    gamma[4] = 0.0;
    gamma[5] = 0.0;

} /* END InitRodas4 */


__global__
void RosenApplyA(size_t ncells32, int istage, double * K, double * Y, double * newY)
{
    size_t idx = blockDim.x*blockIdx.x + threadIdx.x;
    if(idx < ncells32*NVAR) {
        newY[idx] = Y[idx];
        for(int j=0; j<istage; ++j) {
            newY[idx] += d_A[istage*(istage-1)/2+j] * K[ncells32*NVAR*j + idx];
        }
    }
}

__global__
void RosenApplyC(size_t ncells32, int istage, double H, double * K, double * fcn, double * stage)
{
    size_t idx = blockDim.x*blockIdx.x + threadIdx.x;
    if(idx < ncells32*NVAR) {
        stage[idx] = fcn[idx];
        for(int j=0; j<istage; ++j) {
            stage[idx] += (d_C[istage*(istage-1)/2+j] / H) * K[ncells32*NVAR*j + idx];
        }
    }
}

__global__
void RosenApplyM(size_t ncells32, int nstage, double * K, double * Y, double * newY)
{
    size_t idx = blockDim.x*blockIdx.x + threadIdx.x;
    if(idx < ncells32*NVAR) {
        newY[idx] = Y[idx];
        for(int j=0; j<nstage; ++j) {
            newY[idx] += d_M[j] * K[ncells32*NVAR*j + idx];
        }
    }
}

__global__
void RosenErr(size_t ncells32, int nstage, double * K,
        double * Y, double * newY, double * errY,
        int nTol, double * abstol, double * reltol)
{
    size_t idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx < ncells32*NVAR) {
        double err = ZERO;
        for(int j=0; j<nstage; ++j) {
            err += d_E[j] * K[ncells32*NVAR*j + idx];
        }

        int tolidx = (nTol == 1) ? 1 : idx / ncells32;
        double Ymax = fmaxf(fabsf(Y[idx]), fabsf(newY[idx]));
        double scale = abstol[tolidx] + reltol[tolidx] * Ymax;
        errY[idx] = (err * err) / (scale * scale);
    }
}

__global__
void RosenErrNorm(size_t ncells32, double * errY, double * retval)
{
    size_t idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx < ncells32) {
        errY += idx;
        double err = ZERO;
        for (int i=0; i<NVAR; ++i) {
            err += errY[i*ncells32];
        }
        retval[idx] = sqrt(err/(double)NVAR);
    }
}

__global__
void d_RosenStageLHS1(size_t ncells32, double diag, double * jac, double * slhs)
{
    size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < ncells32*JAC_LU_NZ) {
        slhs[idx] = -jac[idx];
    }
}

__global__
void d_RosenStageLHS2(size_t ncells32, double diag, double * jac, double * slhs)
{
    size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < ncells32*NVAR) {
        slhs[(idx % ncells32) + JAC_LU_DIAG[idx/ncells32]*ncells32] += diag;
    }
}

/*----------------------------------------------------------------------------*/
/* Calculates the left hand side matrix for Rosenbrock stage calculation      */
/*                                                                            */
/* @param[in]  diag     Value to add to diagonal elements                     */
/* @param[in]  jac      The Jacobian                                          */
/* @param[out] slhs     Left had side matrix for Rosenbrock stage calculation */
/*----------------------------------------------------------------------------*/
void RosenStageLHS(size_t ncells32, double diag, double * d_jac, double * d_slhs)
{
    size_t nBlocks1 = ((ncells32*JAC_LU_NZ + 255) & ~255) >> 8;
    size_t nBlocks2 = ((ncells32*NVAR + 255) & ~255) >> 8;
    size_t nThreads = 256;
    // Two kernel launches to synchronize between blocks
    d_RosenStageLHS1<<<nBlocks1, nThreads>>>(ncells32, diag, d_jac, d_slhs);
    d_RosenStageLHS2<<<nBlocks2, nThreads>>>(ncells32, diag, d_jac, d_slhs);
} /* END RosenStageLHS */



/*--------------------------------- Integrate ---------------------------------*/
/* Kppa-generated time stepping integrator                                     */
/*                                                                             */
/* @param[in]     ncells Number of grid cells                                  */
/* @param[in,out] d_var  Variable species concentrations in device memory      */
/* @param[in,out] d_fix  Fixed species concentrations in device memory         */
/* @param[in]     idx    Current grid cell index                               */
/* @param[in]     tstart Integration start time                                */
/* @param[in]     tend   Integration end time                                  */
/* @param[in]     abstol Absolute integration tolerances for variable species  */
/* @param[in]     reltol Relative integration tolerances for variable species  */
/* @param[in,out] idata  Integer integration in/out parameters                 */
/* @param[in,out] rdata  Real value integration in/out parameters              */
/*-----------------------------------------------------------------------------*/
void Integrate(size_t const ncells, double d_var[82], double d_fix[1], size_t
    const idx, double const tstart, double const tend, double const
    abstol[82], double const  reltol[82], int idata[20], double rdata[20])
{
    /* .................... Rosenbrock method parameters .................... */

    char * name;        /* Method name */
    int nStage;         /* Number of stages, from 2 to 6 */
    double invLoEst;     /* Inverse local order estimation */
    double M[6];         /* New step solution coefficients */
    double E[6];         /* Error estimation coefficients */
    double alpha[6];     /* Y_stage_i ~ Y( T + H*alpha_i ) */
    double gamma[6];     /* Gamma_i = \sum_j gamma_{i,j} */

    /* Coefficient matrices A and C are strictly lower triangular.
     * The subdiagonal elements are stored in row-wise order:
     * A(2,1)=A[0], A(3,1)=A[1], A(3,2)=A[2], etc. */
    double A[15];
    double C[15];

    /* F[i] == 0: stage i will re-use the function evaluation from stage i-1
     * F[i] != 0: stage i will evaluate function */
    char F[6];

    /* .................... Integration parameters .................... */

    double spanT;        /* Integration time span (positive value) */
    int autonomous;     /* idata[0]: Zero if F = F(t,y) */
    int nTol;           /* idata[1]: Length of the tolerance vectors, 1 = scalar*/
    int stepMax;        /* idata[2]: Maximum permitted steps */
    double minH;         /* rdata[0]: Integration step size lower bound */
    double maxH;         /* rdata[1]: Integration step size upper bound */
    double startH;       /* rdata[2]: Starting integration step size */
    double minFact;      /* rdata[3]: Lower bound on step decrease factor */
    double maxFact;      /* rdata[4]: Upper bound on step increase factor */
    double rejectFact;   /* rdata[5]: Step decrease factor after step rejection */
    double safeFact;     /* rdata[6]: Safety factor in computation of new step size */

    /* .................... Local variables .................... */

    double * d_K = 0;        /* Stage solution vectors */
    double * d_newVar = 0;   /* Variable concentrations after successful solve */
    double * d_errVar = 0;   /* Error in newVar */
    double * d_fcn0 = 0;     /* Function at time tstart */
    double * d_fcn = 0;      /* Function at time T */
    double * d_dFdT = 0;     /* Partial derivative of the function w.r.t T */
    double * d_rct = 0;      /* Reaction rates at time T */
    double * d_jac0 = 0;     /* Jacobian at time tstart */
    double * d_slhs = 0;     /* Stage computation left hand side matrix */
    double * d_abstol = 0;   /* Device memory absolute tolerance vector */
    double * d_reltol = 0;   /* Device memory relative tolerance vector */
    double * d_errNorm = 0;  /* Device memory scaled norms of error vectors */
    double * d_scratch = 0;  /* Device scratch memory */
    double * errNorm = 0;    /* Scaled norms of error vectors for all grid cells */

    int dir;            /* +1 if time advances positively, -1 otherwise */
    double T;            /* Model time */
    double H;            /* Timestep */
    double newH;         /* Updated timestep */
    double errNormMax;   /* Maximum of all scaled norms of error vectors */

    int rejectH;        /* Number of consecutive time step rejections */
    int i;              /* Iterators */

    int nFun = 0;       /* Number of function evaluations */
    int nJac = 0;       /* Number of Jacobian evaluations */
    int nStp = 0;       /* Number of solver steps */
    int nAcc = 0;       /* Number of accepted steps */
    int nRej = 0;       /* Number of rejected steps */
    int nDec = 0;       /* Number of matrix decompositions */
    int nSol = 0;       /* Number of Ax=b solves */
    int nSng = 0;       /* Number of singular decomposition results */

    /* Macro to clean up and abort the integrator */
    #define ABORT(code, fmt, ...) { \
        printf("Kppa: %s: T=%g, H=%g: " fmt, name, T, H, ##__VA_ARGS__); \
        idata[19] = code; \
        goto end; \
    }

    /* ................ Initialize the Rosenbrock method ................ */

    name = "Unknown";
    switch (idata[3]) {
    case 0:
        InitRos4(&name, &nStage, &invLoEst, M, E, A, C, alpha, gamma, F);
        break;
    case 1:
        InitRos2(&name, &nStage, &invLoEst, M, E, A, C, alpha, gamma, F);
        break;
    case 2:
        InitRos3(&name, &nStage, &invLoEst, M, E, A, C, alpha, gamma, F);
        break;
    case 3:
        InitRos4(&name, &nStage, &invLoEst, M, E, A, C, alpha, gamma, F);
        break;
    case 4:
        InitRodas3(&name, &nStage, &invLoEst, M, E, A, C, alpha, gamma, F);
        break;
    case 5:
        InitRodas4(&name, &nStage, &invLoEst, M, E, A, C, alpha, gamma, F);
        break;
    default:
        fprintf(stderr, "Kppa: Unknown method: %d\n", idata[3]);
        idata[19] = -3;
        return;
    }

    /* ................... Initialize local variables ................... */

    /* ncells rounded up to next multiple of 32 */
    size_t ncells32 = (ncells + 31) & ~31;

    /* Size in bytes of the variable concentrations in the integration */
    size_t varsize = ncells32*NVAR*sizeof(double);

    /* Number of threads in each CUDA block */
    size_t nThreads = 256;
    /* Number of blocks in CUDA grid */
    size_t nBlocks = ((ncells32*NVAR + 255) & ~255) >> 8;
    /* Number of blocks for errNorm kernel call */
    size_t errNormBlocks = ((ncells32 + 255) & ~255) >> 8;

    /* Initialize step rejection counter */
    rejectH = 0;

    /* Initialize time */
    dir = (tend >= tstart ? +1 : -1);
    spanT = dir * (tend - tstart);
    T = tstart;
    H = spanT;

    /* Determine if F depends on time */
    autonomous = (idata[0] != 0);

    /* Scalar tolerances limits the tolerance vectors to the first element. */
    nTol = idata[1] ? 1 : NVAR;

    /* Maximum number of steps before the method aborts */
    stepMax = idata[2] ? idata[2] : 100000;
    if (stepMax < 0)
        ABORT(-3, "Invalid maximum steps: %d\n", stepMax);

    /* Check tolerance vectors */
    if(idata[4]) {
        for (i=0; i<nTol; i++) {
            if (abstol[i] <= ZERO)
                ABORT(-3, "Unreasonable tolerance: abstol[%d]=%g\n", i, abstol[i]);
            if (reltol[i] <= (10.0 * DBL_EPSILON) || reltol[i] >= ONE)
                ABORT(-3, "Unreasonable tolerance: reltol[%d]=%g\n", i, reltol[i]);
        }
    }

    /* Lower bound on the step size: (positive value) */
    minH = rdata[0];
    if (minH < ZERO)
        ABORT(-3, "Invalid step size lower bound: %g\n", minH);

    /* Upper bound on the step size: (positive value) */
    maxH = rdata[1] ? fmin(fabs(rdata[1]), spanT) : spanT;
    if (maxH < ZERO)
        ABORT(-3, "Invalid step size upper bound: %g\n", maxH);

    /*  Starting step size: (positive value) */
    startH = rdata[2] ? fmin(fabs(rdata[2]), spanT) : fmax(minH,MIN_DELT);
    if (startH < ZERO)
        ABORT(-3, "Invalid starting step size: %g\n", startH);

    /* Lower bound on step decrease factor */
    minFact = rdata[3] ? rdata[3] : 0.2;
    if (minFact < ZERO)
        ABORT(-3, "Invalid lower bound on step decrease factor: %g\n", minFact);

    /* Upper bound on step increase factor */
    maxFact = rdata[4] ? rdata[4] : 6.0;
    if (maxFact < minFact)
        ABORT(-3, "Invalid upper bound on step increase factor: %g\n", maxFact);

    /* Step decrease factor after step rejection */
    rejectFact = rdata[5] ? rdata[5] : 0.1;
    if (rejectFact < ZERO)
        ABORT(-3, "Invalid step decrease factor for rejected step: %g\n", rejectFact);

    /* Safety factor in the computation of new step size */
    safeFact = rdata[6] ? rdata[6] : 0.9;
    if (safeFact < ZERO)
        ABORT(-3, "Invalid new step safety factor: %g\n", safeFact);

    /* Adjust timestep according to user-specified limits */
    H = fmin(startH, maxH);
    if (fabs(H) < 10 * DBL_EPSILON)
        H = MIN_DELT;
        
    /* Allocate memory */
    if(hipMalloc(&d_K, nStage*varsize) != hipSuccess)
        ABORT(-20, "Can't allocate K on device.\n");
    if(hipMalloc(&d_newVar, varsize) != hipSuccess)
        ABORT(-20, "Can't allocate newVar on device.\n");
    if(hipMalloc(&d_errVar, varsize) != hipSuccess)
        ABORT(-20, "Can't allocate errVar on device.\n");
    if(hipMalloc(&d_fcn0, varsize) != hipSuccess)
        ABORT(-20, "Can't allocate fcn0 on device.\n");
    if(hipMalloc(&d_fcn, varsize) != hipSuccess)
        ABORT(-20, "Can't allocate fcn on device.\n");
    if(hipMalloc(&d_dFdT, varsize) != hipSuccess)
        ABORT(-20, "Can't allocate dFdT on device.\n");
    if(hipMalloc(&d_rct, ncells32*NREACT*sizeof(double)) != hipSuccess)
        ABORT(-20, "Can't allocate rct on device.\n");
    if(hipMalloc(&d_jac0, ncells32*JAC_LU_NZ*sizeof(double)) != hipSuccess)
        ABORT(-20, "Can't allocate jac0 on device.\n");
    if(hipMalloc(&d_slhs, ncells32*JAC_LU_NZ*sizeof(double)) != hipSuccess)
        ABORT(-20, "Can't allocate slhs on device.\n");
    if(hipMalloc(&d_abstol, NVAR*sizeof(double)) != hipSuccess)
        ABORT(-20, "Can't allocate abstol on device.\n");
    if(hipMalloc(&d_reltol, NVAR*sizeof(double)) != hipSuccess)
        ABORT(-20, "Can't allocate reltol on device.\n");
    if(hipMalloc(&d_errNorm, ncells32*sizeof(double)) != hipSuccess)
        ABORT(-20, "Can't allocate errNorm on device.\n");
    if(hipMalloc(&d_scratch, ncells32*JAC_LU_NZ*sizeof(double)) != hipSuccess)
        ABORT(-20, "Can't allocate scratch on device.\n");
    if(hipHostMalloc(&errNorm, ncells32*sizeof(double)) != hipSuccess)
        ABORT(-20, "Can't allocate paged locked errNorm on host.\n");

    /* Copy tolerance vectors to device */
    if(nTol > 1) {
        if(hipMemcpy(d_abstol, abstol, NVAR*sizeof(double), hipMemcpyHostToDevice) != hipSuccess)
            ABORT(-20, "Can't copy absolute tolerance vector to device.\n");
        if(hipMemcpy(d_reltol, reltol, NVAR*sizeof(double), hipMemcpyHostToDevice) != hipSuccess)
            ABORT(-20, "Can't copy relative tolerance vector to device.\n");
    }

    /* Copy Rosenbrock method parameters to device constant memory */
    if(hipMemcpyToSymbol(HIP_SYMBOL(d_M), M, sizeof(M), 0, hipMemcpyHostToDevice) != hipSuccess)
        ABORT(-20, "Can't copy new step solution coefficients to device.\n");
    if(hipMemcpyToSymbol(HIP_SYMBOL(d_E), E, sizeof(E), 0, hipMemcpyHostToDevice) != hipSuccess)
        ABORT(-20, "Can't copy error estimation coefficients to device.\n");
    if(hipMemcpyToSymbol(HIP_SYMBOL(d_A), A, sizeof(A), 0, hipMemcpyHostToDevice) != hipSuccess)
        ABORT(-20, "Can't copy coefficient matrix A to device.\n");
    if(hipMemcpyToSymbol(HIP_SYMBOL(d_C), C, sizeof(C), 0, hipMemcpyHostToDevice) != hipSuccess)
        ABORT(-20, "Can't copy coefficient matrix C to device.\n");

    /* ............................ Integrate ............................ */

    while(fabs(tend - T) > DBL_EPSILON) {   /* Time integration loop */

        /* Check step count */
        if (nStp > stepMax)
            ABORT(-6, "Too many integration steps: stepMax=%d\n", stepMax);

        /* Check timestep size */
        if ((T + 0.1*H == T) || (H <= DBL_EPSILON))
            ABORT(-7, "Step size too small (T + H/10 = T) or H < eps\n");

        /* Update timestep */
        H = fmin(H,fabs(tend-T));

        /* Compute reaction rates at the current time */
        Rates(ncells32, T, idx, d_rct);

        /* Compute the function at the current time */
        Fun(ncells32, d_var, d_fix, d_rct, d_fcn0, d_scratch);
        ++nFun;

        /* Compute the Jacobian at the current time */
        Jac(ncells32, d_var, d_fix, d_rct, d_jac0, d_scratch);
        ++nJac;

        /* Compute the function derivative with respect to time */
        if (!autonomous) {
            double delta = sqrt(FLT_EPSILON) * fmax(MIN_DELT, fabs(T));
            Rates(ncells32, T+delta, idx, d_rct);
            Fun(ncells32, d_var, d_fix, d_rct, d_fcn, d_scratch);
            ++nFun;
            WYMXDA(ncells32, NVAR, d_fcn0, d_fcn, delta, d_dFdT);
        }

        /* Repeat step calculation until step accepted  */
        do {
            int singRow = 0;
            int decomps = 1;

            /* Prepare the LHS matrix for stage calculations */
            RosenStageLHS(ncells32, 1.0/(dir*H*gamma[0]), d_jac0, d_slhs);

            /* LU decompose stage LHS matrix */
            singRow = Decomp(ncells32, d_slhs);
            ++nDec;

            /* If the decomposition failed, half the timestep and try again */
            while(singRow) {
                printf("Kppa: %s: LU decomposition singular on row %d\n", name, singRow-1);
                ++nSng;

                /* Reduce step size */
                H *= HALF;

                /* Abort after eight failed decompositions */
                if (decomps > 8 || H == ZERO)
                    ABORT(-8, "Matrix is repeatedly singular\n");

                /* Build new stage LHS with reduced time step */
                RosenStageLHS(ncells32, 1.0/(dir*H*gamma[0]), d_jac0, d_slhs);

                /* LU decompose stage LHS matrix */
                singRow = Decomp(ncells32, d_slhs);
                ++nDec;
                ++decomps;
            }

            /* Compute stage 0 using the previously-computed function */
            WCOPY(ncells32, NVAR, d_fcn0, d_fcn);
            WCOPY(ncells32, NVAR, d_fcn0, d_K);
            if (!autonomous && gamma[0]) {
                WAXPY(ncells32, NVAR, dir*H*gamma[0], d_dFdT, d_K);
            }

            /* Solve stage 0 */
            Solve(ncells32, d_slhs, d_K);
            nSol++;

            /* Compute the remaining stages  */
            for (int i=1; i<nStage; ++i) {
                double * d_stage = d_K + ncells32 * NVAR * i;

                if (F[i]) {
                    double tau = T + alpha[i] * dir * H;

                    /* Apply coefficient matrix A */
                    RosenApplyA<<<nBlocks, nThreads>>>(ncells32, i, d_K, d_var, d_newVar);

                    /* Update reaction rates, if necessary */
                    if(!autonomous) {
                        Rates(ncells32, tau, idx, d_rct);
                    }

                    /* Evaluate the function */
                    Fun(ncells32, d_newVar, d_fix, d_rct, d_fcn, d_scratch);
                    ++nFun;
                }

                /* Apply coefficient matrix C */
                RosenApplyC<<<nBlocks, nThreads>>>(ncells32, i, dir*H, d_K, d_fcn, d_stage);

                if (!autonomous && gamma[i]) {
                    WAXPY(ncells32, NVAR, dir*H*gamma[i], d_dFdT, d_stage);
                }

                /* Solve stage i */
                Solve(ncells32, d_slhs, d_stage);
                nSol++;
            }

            /* Compute the new solution */
            RosenApplyM<<<nBlocks, nThreads>>>(ncells32, nStage, d_K, d_var, d_newVar);

            /* Calculate error vectors */
            RosenErr<<<nBlocks, nThreads>>>(ncells32, nStage, d_K, d_var, d_newVar, d_errVar, nTol, d_abstol, d_reltol);

            /* Calculate the scaled norm of the error vector in each grid cell */
            RosenErrNorm<<<errNormBlocks, nThreads>>>(ncells32, d_errVar, d_errNorm);

            /* Find the largest scaled norm */
            hipMemcpy(errNorm, d_errNorm, ncells32*sizeof(double), hipMemcpyDeviceToHost);
            errNormMax = 0;
            for(int i=0; i<ncells; ++i) {
                double nrm = errNorm[i];
                if(isinf(nrm)) {
                    ABORT(-10, "Error norm in cell %zu is Inf\n", idx);
                } else if(isnan(nrm)) {
                    ABORT(-10, "Error norm in cell %zu is NaN\n", idx);
                }
                errNormMax = fmaxf(errNormMax, nrm);
            }
            rdata[12] = errNormMax;

            /* Calculate a new step size: minFact <= newH/H <= maxFact */
            newH = H * fmin(maxFact,fmax(minFact,safeFact/pow(errNormMax,invLoEst)));
            ++nStp;

            /* Decide to accept or reject step  */
            if (errNormMax <= ONE || H <= minH) {
                /* Step accepted */
                ++nAcc;
                WCOPY(ncells32, NVAR, d_newVar, d_var);
                T += dir * H;
                /* Adjust step size */
                newH = fmax(minH,fmin(newH,maxH));
                if(rejectH) {
                    newH = fmin(newH,H);
                }
                rejectH = 0;
                H = newH;
                /* Return to time loop */
                break;
            } else {
                /* Step rejected */
                ++nRej;
                if(rejectH > 1) {
                    newH = H * rejectFact;
                }
                ++rejectH;
                H = newH;
                /* Continue step calculation */
                continue;
            }

        } while(1); /* Step calculation */

    } /* Time loop */

    /* ...................... Exit integrator ...................... */
    
    /* Set exit status */
    idata[19] = 0;

end:

    /* Deallocate memory */
    hipFree((void*)d_K);
    hipFree((void*)d_newVar);
    hipFree((void*)d_errVar);
    hipFree((void*)d_fcn0);
    hipFree((void*)d_fcn);
    hipFree((void*)d_rct);
    hipFree((void*)d_dFdT);
    hipFree((void*)d_jac0);
    hipFree((void*)d_slhs);
    hipFree((void*)d_abstol);
    hipFree((void*)d_reltol);
    hipFree((void*)d_errNorm);
    hipFree((void*)d_scratch);
    hipHostFree((void*)errNorm);

    /* Collect statistics */
    idata[10] = nFun;
    idata[11] = nJac;
    idata[12] = nStp;
    idata[13] = nAcc;
    idata[14] = nRej;
    idata[15] = nDec;
    idata[16] = nSol;
    idata[17] = nSng;
    /* Record exit time and last step size */
    rdata[10] = T;
    rdata[11] = H;
    rdata[12] = errNormMax;

}/* END Integrate */



/*----------------------- END kppa_CUDA_rosenbrock.h END ----------------------*/
